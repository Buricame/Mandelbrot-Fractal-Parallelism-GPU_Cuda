﻿
#include <iostream>
#include <fstream>
#include <complex>

#include <hip/hip_runtime.h>


// '<<<' operatörü visual studioda çalışmadığı için makro atama yapıldı
#ifndef __INTELLISENSE__
#define KERNEL_ARGS2(grid, block)                 <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem)         <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

// Yükseklik genişlik ve iterasyon sayısını ayarlama
#define WIDTH 100000
#define HEIGHT 100000
#define MAX_ITER 1000

// CUDA için kernelda global değişkeni kullanılarak mandelbrot hesaplanmıştır
__global__ void mandelbrot(int* output) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT) {
        double dx = 3.0 / WIDTH;
        double dy = 2.0 / HEIGHT;

        double x0 = x * dx - 2.0;
        double y0 = y * dy - 1.0;

        double zx = 0.0;
        double zy = 0.0;
        double zx2 = 0.0;
        double zy2 = 0.0;

        int iter = 0;
        while (zx2 + zy2 < 4 && iter < MAX_ITER) {
            zy = 2 * zx * zy + y0;
            zx = zx2 - zy2 + x0;
            zx2 = zx * zx;
            zy2 = zy * zy;
            iter++;
        }

        output[y * WIDTH + x] = iter;
    }
}

// Çıktıyı bmp resim dosyası olarak kaydeder
void saveBMP(int* data, const char* filename) {
    std::ofstream outFile(filename, std::ios::out | std::ios::binary);

    // BMP header
    unsigned char header[54] = {
        0x42, 0x4D, 0, 0, 0, 0, 0, 0, 0, 0, 54, 0, 0, 0, 40, 0,
        0, 0, (unsigned char)(WIDTH & 0xff), (unsigned char)((WIDTH >> 8) & 0xff), (unsigned char)((WIDTH >> 16) & 0xff), (unsigned char)((WIDTH >> 24) & 0xff),
        (unsigned char)(HEIGHT & 0xff), (unsigned char)((HEIGHT >> 8) & 0xff), (unsigned char)((HEIGHT >> 16) & 0xff), (unsigned char)((HEIGHT >> 24) & 0xff),
        1, 0, 24, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
    };

    // Write header
    outFile.write(reinterpret_cast<const char*>(header), sizeof(header));

    // Write image data
    for (int y = HEIGHT - 1; y >= 0; --y) {
        for (int x = 0; x < WIDTH; ++x) {
            unsigned char color = (data[y * WIDTH + x] == MAX_ITER) ? 0 : 255;
            outFile.write(reinterpret_cast<const char*>(&color), sizeof(color));
            outFile.write(reinterpret_cast<const char*>(&color), sizeof(color));
            outFile.write(reinterpret_cast<const char*>(&color), sizeof(color));
        }
        // Padding to make sure each row's size is a multiple of 4
        const unsigned char pad = 0;
        for (int p = 0; p < (4 - (WIDTH * 3) % 4) % 4; ++p) {
            outFile.write(reinterpret_cast<const char*>(&pad), sizeof(pad));
        }
    }

    outFile.close();
}

int main() {
    int* output;
    hipMallocManaged(&output, WIDTH * HEIGHT * sizeof(int));

    //Cuda thread yapılandırması belirlenir.(Yükseklik,Genişlik)
    dim3 block(16, 16);
    dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);

    //Hesaplama süresini hesaplamak için başlangıç ve bitiş süreleri tanımlanır ve tutulur
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    mandelbrot KERNEL_ARGS2(grid, block) (output); // Cuda kerneldeki kodu çalıştırır.
    hipDeviceSynchronize(); //Hesaplama yaparken Cuda çekirdekleri arasında senkronizasyonu sağlar

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Hesaplama süresini miliseconds float olarak hesaplayan kod
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;

    saveBMP(output, "mandelbrot.bmp"); // Resmi bmp formatı ile diske bu isimle kaydeder

    hipFree(output); // Cudaları serbest bırakır ve işlemi sonlandırır

    return 0;
}
